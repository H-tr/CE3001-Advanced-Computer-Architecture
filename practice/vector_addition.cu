#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ 
void vector_add_cu(int* d_c, int* d_a, int* d_b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    d_c[i] = d_a[i] + d_b[i];
}

int main(void) {
    int N = 3;
    int a[N] = {7, 2, 3};
    int b[N] = {6, 4, 5};
    int c[N];

    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, sizeof(int)*N);
    hipMalloc((void**)&d_b, sizeof(int)*N);
    hipMalloc((void**)&d_c, sizeof(int)*N);

    hipMemcpy(d_a, a, sizeof(int)*N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(int)*N, hipMemcpyHostToDevice);

    vector_add_cu<<<1, 3>>>(d_c, d_a, d_b);
    hipDeviceSynchronize();

    hipMemcpy(c, d_c, sizeof(int)*N, hipMemcpyDeviceToHost);
    for (int i = 0; i < N; ++i)
        printf("%d ", c[i]);
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}