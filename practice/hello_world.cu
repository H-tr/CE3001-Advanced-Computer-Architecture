// multi dimensional threads

#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void hello_GPU(void) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    printf("Hello from GPU[%d][%d]!\n", i, j);
}

int main() {
    #define BLOCK_SIZE 16
    #define GRID_SIZE 1
    
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);  // now the threads are BLOCK_SIZE*BLOCK_SIZE
    dim3 dimGrid(GRID_SIZE, GRID_SIZE);     // 1 * 1 blocks in a grid
    hello_GPU<<<dimGrid, dimBlock>>>();
    hipDeviceSynchronize();
    printf("Hello from CPU!\n");
    return 0;
}