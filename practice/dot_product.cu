
#include <hip/hip_runtime.h>
#include <stdio.h>

#define SIZE 3

__global__
void dot_prod_cu(int* d_c, int* d_a, int* d_b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ int temp[SIZE];
    temp[i] = d_a[i] * d_b[i];
    __syncthreads();
    if (i == 0) {
        int sum = 0;
        for (int j = 0; j < SIZE; ++j)
            sum += temp[j];
        *d_c = sum;
    }
}

int main(void) {
    int a[SIZE] = {1, 2, 3};
    int b[SIZE] = {4, 5, 6};
    int c;

    int *d_a, *d_b, *d_c;
    hipMalloc((void**) &d_a, sizeof(int) * SIZE);
    hipMalloc((void**) &d_b, sizeof(int) * SIZE);
    hipMalloc((void**) &d_c, sizeof(int));

    hipMemcpy(d_a, a, sizeof(int)*SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(int)*SIZE, hipMemcpyHostToDevice);
    dot_prod_cu<<<1, 3>>>(d_c, d_a, d_b);
    hipDeviceSynchronize();

    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);
    printf("The number of dotprod is: %d\n", c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}